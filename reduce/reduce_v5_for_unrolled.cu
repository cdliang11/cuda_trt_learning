
#include <hip/hip_runtime.h>

#include <time.h>

#define N 32*1024*1024    // 输入数据的长度
#define BLOCK_SIZE 256    // 每个block的线程数，也就是一个block有8个wrap，每个block要计算的元素个数是256，一个warp有32个线程


template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache, int tid) {
  if (blockSize >= 64) cache[tid] += cache[tid + 32];
  if (blockSize >= 32) cache[tid] += cache[tid + 16];
  if (blockSize >= 16) cache[tid] += cache[tid + 8];
  if (blockSize >= 8) cache[tid] += cache[tid + 4];
  if (blockSize >= 4) cache[tid] += cache[tid + 2];
  if (blockSize >= 2) cache[tid] += cache[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce_kernel_v5_for_unrolled(float *g_idata, float *g_odata) {
  // 申请共享内存
  __shared__ float sdata[BLOCK_SIZE];

  // 每个线程从全局内存中读取一个数据到共享内存
  unsigned int tid = threadIdx.x;   // 线程id
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;  // 全局id
  // sdata[tid] = g_idata[i];  // 从全局内存中读取数据到共享内存
  sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];  // 从全局内存中读取数据到共享内存 一个线程读取两个数据

  __syncthreads();   // 同步

//   for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
//     if (tid < s) {
//       sdata[tid] += sdata[tid + s];
//     }
//     __syncthreads();  // 同步
//   }

  if (blockSize >= 512) {
    if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
  }
  if (blockSize >= 256) {
    if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
  }
  if (blockSize >= 128) {
    if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
  }

  if (tid < 32) warpReduce<blockSize>(sdata, tid);

  // 写结果到全局内存
  if (tid == 0) {
    g_odata[blockIdx.x] = sdata[0];
  }

}

int main() {
  float *input_host = (float *)malloc(N * sizeof(float));
  float *input_device;
  hipMalloc((void **)&input_device, N * sizeof(float));  //  申请显存
  for (int i = 0; i < N; i++) {
    input_host[i] = 2.0;
  }
  hipMemcpy(input_device, input_host, N * sizeof(float), hipMemcpyHostToDevice);  // 从主机内存拷贝到显存

  int32_t block_num = (N + BLOCK_SIZE - 1) / BLOCK_SIZE / 2;
  float *output_host = (float *)malloc((N / BLOCK_SIZE) * sizeof(float));
  float *output_device;
  hipMalloc((void **)&output_device, (N / BLOCK_SIZE) * sizeof(float));  //  申请显存

  dim3 grid(N / BLOCK_SIZE, 1);   // 会自动在最后边补1  grid(N/BLOCK_SIZE, 1, 1)
  dim3 block(BLOCK_SIZE, 1);
  reduce_kernel_v5_for_unrolled<BLOCK_SIZE><<<grid, block>>>(input_device, output_device);
  hipMemcpy(output_device, output_host, block_num * sizeof(float), hipMemcpyDeviceToHost);  // 从显存拷贝到主机内存
  return 0;
}
