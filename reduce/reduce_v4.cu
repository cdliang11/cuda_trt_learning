
#include <hip/hip_runtime.h>

#include <time.h>

#define N 32*1024*1024    // 输入数据的长度
#define BLOCK_SIZE 256    // 每个block的线程数，也就是一个block有8个wrap，每个block要计算的元素个数是256，一个warp有32个线程


__device__ void warpReduce(volatile float *sdata, int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

__global__ void reduce_kernel_v4(float *g_idata, float *g_odata) {
  // 申请共享内存
  __shared__ float sdata[BLOCK_SIZE];

  // 每个线程从全局内存中读取一个数据到共享内存
  unsigned int tid = threadIdx.x;   // 线程id
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;  // 全局id
  // sdata[tid] = g_idata[i];  // 从全局内存中读取数据到共享内存
  sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];  // 从全局内存中读取数据到共享内存 一个线程读取两个数据

  __syncthreads();   // 同步

  // 在共享内存上做reduce
  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();  // 同步
  }
  if (tid < 32) warpReduce(sdata, tid);

  // 写结果到全局内存
  if (tid == 0) {
    g_odata[blockIdx.x] = sdata[0];
  }

}

int main() {
  float *input_host = (float *)malloc(N * sizeof(float));
  float *input_device;
  hipMalloc((void **)&input_device, N * sizeof(float));  //  申请显存
  for (int i = 0; i < N; i++) {
    input_host[i] = 2.0;
  }
  hipMemcpy(input_device, input_host, N * sizeof(float), hipMemcpyHostToDevice);  // 从主机内存拷贝到显存

  int32_t block_num = (N + BLOCK_SIZE - 1) / BLOCK_SIZE / 2;
  float *output_host = (float *)malloc((N / BLOCK_SIZE) * sizeof(float));
  float *output_device;
  hipMalloc((void **)&output_device, (N / BLOCK_SIZE) * sizeof(float));  //  申请显存

  dim3 grid(N / BLOCK_SIZE, 1);   // 会自动在最后边补1  grid(N/BLOCK_SIZE, 1, 1)
  dim3 block(BLOCK_SIZE, 1);
  reduce_kernel_v4<<<grid, block>>>(input_device, output_device);
  hipMemcpy(output_device, output_host, block_num * sizeof(float), hipMemcpyDeviceToHost);  // 从显存拷贝到主机内存
  return 0;
}
